#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "sa_cuda_context.hpp"

#include <stdio.h>
#include <cstdint>
#include <chrono>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <functional>
#include <vector>
#include <iostream>
#include <stdint.h>


//*blake header */

typedef struct  blake2b_state_s
{
	uint64_t    h[8];
	uint64_t    bytes;
}               blake2b_state_t;

void zcash_blake2b_init(blake2b_state_t *st, uint8_t hash_len, uint32_t n, uint32_t k);
void zcash_blake2b_update(blake2b_state_t *st, const uint8_t *_msg,
	uint32_t msg_len, uint32_t is_final);
void zcash_blake2b_final(blake2b_state_t *st, uint8_t *out, uint8_t outlen);

/* blake.cpp **/

//static const uint32_t   blake2b_block_len = 128;
static const uint32_t   blake2b_rounds = 12;
static const uint64_t   blake2b_iv[8] =
{
	0x6a09e667f3bcc908ULL, 0xbb67ae8584caa73bULL,
	0x3c6ef372fe94f82bULL, 0xa54ff53a5f1d36f1ULL,
	0x510e527fade682d1ULL, 0x9b05688c2b3e6c1fULL,
	0x1f83d9abfb41bd6bULL, 0x5be0cd19137e2179ULL,
};
static const uint8_t    blake2b_sigma[12][16] =
{
	{ 0,  1,  2,  3,  4,  5,  6,  7,  8,  9, 10, 11, 12, 13, 14, 15 },
	{ 14, 10,  4,  8,  9, 15, 13,  6,  1, 12,  0,  2, 11,  7,  5,  3 },
	{ 11,  8, 12,  0,  5,  2, 15, 13, 10, 14,  3,  6,  7,  1,  9,  4 },
	{ 7,  9,  3,  1, 13, 12, 11, 14,  2,  6,  5, 10,  4,  0, 15,  8 },
	{ 9,  0,  5,  7,  2,  4, 10, 15, 14,  1, 11, 12,  6,  8,  3, 13 },
	{ 2, 12,  6, 10,  0, 11,  8,  3,  4, 13,  7,  5, 15, 14,  1,  9 },
	{ 12,  5,  1, 15, 14, 13,  4, 10,  0,  7,  6,  3,  9,  2,  8, 11 },
	{ 13, 11,  7, 14, 12,  1,  3,  9,  5,  0, 15,  4,  8,  6,  2, 10 },
	{ 6, 15, 14,  9, 11,  3,  0,  8, 12,  2, 13,  7,  1,  4, 10,  5 },
	{ 10,  2,  8,  4,  7,  6,  1,  5, 15, 11,  9, 14,  3, 12, 13,  0 },
	{ 0,  1,  2,  3,  4,  5,  6,  7,  8,  9, 10, 11, 12, 13, 14, 15 },
	{ 14, 10,  4,  8,  9, 15, 13,  6,  1, 12,  0,  2, 11,  7,  5,  3 },
};

/*
** Init the state according to Zcash parameters.
*/
void zcash_blake2b_init(blake2b_state_t *st, uint8_t hash_len,
	uint32_t n, uint32_t k)
{
	st->h[0] = blake2b_iv[0] ^ (0x01010000 | hash_len);
	for (uint32_t i = 1; i <= 5; i++)
		st->h[i] = blake2b_iv[i];
	st->h[6] = blake2b_iv[6] ^ *(uint64_t *)"ZcashPoW";
	st->h[7] = blake2b_iv[7] ^ (((uint64_t)k << 32) | n);
	st->bytes = 0;
}

static uint64_t rotr64(uint64_t a, uint8_t bits)
{
	return (a >> bits) | (a << (64 - bits));
}

static inline void mix64(uint64_t *va, uint64_t *vb, uint64_t *vc, uint64_t *vd,
	uint64_t x, uint64_t y)
{
	*va = (*va + *vb + x);
	*vd = rotr64(*vd ^ *va, 32);
	*vc = (*vc + *vd);
	*vb = rotr64(*vb ^ *vc, 24);
	*va = (*va + *vb + y);
	*vd = rotr64(*vd ^ *va, 16);
	*vc = (*vc + *vd);
	*vb = rotr64(*vb ^ *vc, 63);
}

/*
** Process either a full message block or the final partial block.
** Note that v[13] is not XOR'd because st->bytes is assumed to never overflow.
**
** _msg         pointer to message (must be zero-padded to 128 bytes if final block)
** msg_len      must be 128 (<= 128 allowed only for final partial block)
** is_final     indicate if this is the final block
*/
void zcash_blake2b_update(blake2b_state_t *st, const uint8_t *_msg,
	uint32_t msg_len, uint32_t is_final)
{
	const uint64_t      *m = (const uint64_t *)_msg;
	uint64_t            v[16];
	memcpy(v + 0, st->h, 8 * sizeof(*v));
	memcpy(v + 8, blake2b_iv, 8 * sizeof(*v));
	v[12] ^= (st->bytes += msg_len);
	v[14] ^= is_final ? -1 : 0;
	for (uint32_t round = 0; round < blake2b_rounds; round++)
	{
		const uint8_t   *s = blake2b_sigma[round];
		mix64(v + 0, v + 4, v + 8, v + 12, m[s[0]], m[s[1]]);
		mix64(v + 1, v + 5, v + 9, v + 13, m[s[2]], m[s[3]]);
		mix64(v + 2, v + 6, v + 10, v + 14, m[s[4]], m[s[5]]);
		mix64(v + 3, v + 7, v + 11, v + 15, m[s[6]], m[s[7]]);
		mix64(v + 0, v + 5, v + 10, v + 15, m[s[8]], m[s[9]]);
		mix64(v + 1, v + 6, v + 11, v + 12, m[s[10]], m[s[11]]);
		mix64(v + 2, v + 7, v + 8, v + 13, m[s[12]], m[s[13]]);
		mix64(v + 3, v + 4, v + 9, v + 14, m[s[14]], m[s[15]]);
	}
	for (uint32_t i = 0; i < 8; i++)
		st->h[i] ^= v[i] ^ v[i + 8];
}

void zcash_blake2b_final(blake2b_state_t *st, uint8_t *out, uint8_t outlen)
{
	memcpy(out, st->h, outlen);
}

/* end of blake cpp*/

#define PARAM_N 200
#define PARAM_K 9
#define PREFIX (PARAM_N / (PARAM_K + 1))
#define NR_INPUTS (1 << PREFIX);
// Approximate log base 2 of number of elements in hash tables
#define APX_NR_ELMS_LOG (PREFIX + 1)

#define ZCASH_BLOCK_HEADER_LEN		140
#define NR_ROWS_LOG 20
#define OVERHEAD 6
#define NR_ROWS (1 << NR_ROWS_LOG)
#define NR_SLOTS ((1 << (APX_NR_ELMS_LOG - NR_ROWS_LOG)) * OVERHEAD)
// Length of 1 element (slot) in bytes
#define SLOT_LEN 32
#define ZCASH_HASH_LEN  50
#define COLL_DATA_SIZE_PER_TH		(NR_SLOTS * 5)
#define MAX_SOLS 10

#define xi_offset_for_round(round)	(8 + ((round) / 2) * 4)


constexpr uint32_t c_NR_SLOTS = NR_SLOTS;
constexpr uint32_t c_ROW_LEN = c_NR_SLOTS * SLOT_LEN;
//constexpr uint32_t c_NR_ROWS = NR_ROWS;

#define HT_SIZE				(NR_ROWS * NR_SLOTS * SLOT_LEN)

#define WN PARAM_N
#define WK PARAM_K

#define COLLISION_BIT_LENGTH (WN / (WK+1))
#define COLLISION_BYTE_LENGTH ((COLLISION_BIT_LENGTH+7)/8)
#define FINAL_FULL_WIDTH (2*COLLISION_BYTE_LENGTH+sizeof(uint32_t)*(1 << (WK)))

#define NDIGITS   (WK+1)
#define DIGITBITS (WN/(NDIGITS))
#define PROOFSIZE (1u<<WK)
#define COMPRESSED_PROOFSIZE ((COLLISION_BIT_LENGTH+1)*PROOFSIZE*4/(8*sizeof(uint32_t)))


typedef struct __align__(64) sols_s
{
	uint32_t nr;
	uint32_t likely_invalids;
	uint8_t valid[MAX_SOLS];
	uint32_t values[MAX_SOLS][(1 << PARAM_K)];
} sols_t;


__device__ uint32_t rowCounter0[1 << NR_ROWS_LOG];
__device__ uint32_t rowCounter1[1 << NR_ROWS_LOG];
__device__ uint32_t* rowCounters[2] = { rowCounter0 , rowCounter1 };
__device__ blake2b_state_t blake;
__device__ sols_t sols;


__constant__ uint64_t blake_iv[] =
{
	0x6a09e667f3bcc908, 0xbb67ae8584caa73b,
	0x3c6ef372fe94f82b, 0xa54ff53a5f1d36f1,
	0x510e527fade682d1, 0x9b05688c2b3e6c1f,
	0x1f83d9abfb41bd6b, 0x5be0cd19137e2179,
};


__global__ void kernel_init_0()
{
	rowCounter0[(blockDim.x * blockIdx.x) + threadIdx.x] = 0;
}

__global__ void kernel_init_1()
{
	rowCounter1[(blockDim.x * blockIdx.x) + threadIdx.x] = 0;
}


typedef uint64_t ulong;
typedef uint32_t uint;
typedef uint8_t uchar;

__device__ uint ht_store(uint round, char *ht, uint i,
	ulong xi0, ulong xi1, ulong xi2, ulong xi3, uint *rowCounters)
{
	uint    row;
	char       *p;
	uint                cnt;
#if NR_ROWS_LOG == 16
	if (!(round & 1))
		row = (xi0 & 0xffff);
	else
		// if we have in hex: "ab cd ef..." (little endian xi0) then this
		// formula computes the row as 0xdebc. it skips the 'a' nibble as it
		// is part of the PREFIX. The Xi will be stored starting with "ef...";
		// 'e' will be considered padding and 'f' is part of the current PREFIX
		row = ((xi0 & 0xf00) << 4) | ((xi0 & 0xf00000) >> 12) |
		((xi0 & 0xf) << 4) | ((xi0 & 0xf000) >> 12);
#elif NR_ROWS_LOG == 18
	if (!(round & 1))
		row = (xi0 & 0xffff) | ((xi0 & 0xc00000) >> 6);
	else
		row = ((xi0 & 0xc0000) >> 2) |
		((xi0 & 0xf00) << 4) | ((xi0 & 0xf00000) >> 12) |
		((xi0 & 0xf) << 4) | ((xi0 & 0xf000) >> 12);
#elif NR_ROWS_LOG == 19
	if (!(round & 1))
		row = (xi0 & 0xffff) | ((xi0 & 0xe00000) >> 5);
	else
		row = ((xi0 & 0xe0000) >> 1) |
		((xi0 & 0xf00) << 4) | ((xi0 & 0xf00000) >> 12) |
		((xi0 & 0xf) << 4) | ((xi0 & 0xf000) >> 12);
#elif NR_ROWS_LOG == 20
	if (!(round & 1))
		row = (xi0 & 0xffff) | ((xi0 & 0xf00000) >> 4);
	else
		row = ((xi0 & 0xf0000) >> 0) |
		((xi0 & 0xf00) << 4) | ((xi0 & 0xf00000) >> 12) |
		((xi0 & 0xf) << 4) | ((xi0 & 0xf000) >> 12);
#else
#error "unsupported NR_ROWS_LOG"
#endif
	xi0 = (xi0 >> 16) | (xi1 << (64 - 16));
	xi1 = (xi1 >> 16) | (xi2 << (64 - 16));
	xi2 = (xi2 >> 16) | (xi3 << (64 - 16));
	cnt = atomicAdd(&rowCounters[row], 1);
	if (cnt >= c_NR_SLOTS) {
		// avoid overflows
		atomicSub(&rowCounters[row], 1);
		return 1;
	}
	p = ht + row * c_ROW_LEN;
	p += cnt * SLOT_LEN + xi_offset_for_round(round);
	// store "i" (always 4 bytes before Xi)
	*(uint *)(p - 4) = i;
	if (round == 0 || round == 1)
	{
		// store 24 bytes
		*(ulong *)(p + 0) = xi0;
		*(ulong *)(p + 8) = xi1;
		*(ulong *)(p + 16) = xi2;
	}
	else if (round == 2)
	{
		// store 20 bytes
		*(uint *)(p + 0) = xi0;
		*(ulong *)(p + 4) = (xi0 >> 32) | (xi1 << 32);
		*(ulong *)(p + 12) = (xi1 >> 32) | (xi2 << 32);
	}
	else if (round == 3)
	{
		// store 16 bytes
		*(uint *)(p + 0) = xi0;
		*(ulong *)(p + 4) = (xi0 >> 32) | (xi1 << 32);
		*(uint *)(p + 12) = (xi1 >> 32);
	}
	else if (round == 4)
	{
		// store 16 bytes
		*(ulong *)(p + 0) = xi0;
		*(ulong *)(p + 8) = xi1;
	}
	else if (round == 5)
	{
		// store 12 bytes
		*(ulong *)(p + 0) = xi0;
		*(uint *)(p + 8) = xi1;
	}
	else if (round == 6 || round == 7)
	{
		// store 8 bytes
		*(uint *)(p + 0) = xi0;
		*(uint *)(p + 4) = (xi0 >> 32);
	}
	else if (round == 8)
	{
		// store 4 bytes
		*(uint *)(p + 0) = xi0;
	}
	return 0;
}

#define rotate(a, bits) ((a) << (bits)) | ((a) >> (64 - (bits)))

#define mix(va, vb, vc, vd, x, y) \
    va = (va + vb + x); \
vd = rotate((vd ^ va), (ulong)64 - 32); \
vc = (vc + vd); \
vb = rotate((vb ^ vc), (ulong)64 - 24); \
va = (va + vb + y); \
vd = rotate((vd ^ va), (ulong)64 - 16); \
vc = (vc + vd); \
vb = rotate((vb ^ vc), (ulong)64 - 63);

__global__
void kernel_round0(char *ht, uint32_t inputs_per_thread)
{
	typedef uint64_t ulong;

	uint32_t                tid = blockIdx.x * blockDim.x + threadIdx.x;
	uint64_t				v[16];
	//uint32_t                inputs_per_thread = c_NR_ROWS / (gridDim.x * blockDim.x);
	uint32_t                input = tid * inputs_per_thread;
	uint32_t                input_end = (tid + 1) * inputs_per_thread;
	uint32_t                dropped = 0;

	while (input < input_end) {
		// shift "i" to occupy the high 32 bits of the second ulong word in the
		// message block
		ulong word1 = (ulong)input << 32;
		// init vector v
		v[0] = blake.h[0];
		v[1] = blake.h[1];
		v[2] = blake.h[2];
		v[3] = blake.h[3];
		v[4] = blake.h[4];
		v[5] = blake.h[5];
		v[6] = blake.h[6];
		v[7] = blake.h[7];
		v[8] = blake_iv[0];
		v[9] = blake_iv[1];
		v[10] = blake_iv[2];
		v[11] = blake_iv[3];
		v[12] = blake_iv[4];
		v[13] = blake_iv[5];
		v[14] = blake_iv[6];
		v[15] = blake_iv[7];
		// mix in length of data
		v[12] ^= ZCASH_BLOCK_HEADER_LEN + 4 /* length of "i" */;
		// last block
		v[14] ^= (ulong)-1;

		// round 1
		mix(v[0], v[4], v[8], v[12], 0, word1);
		mix(v[1], v[5], v[9], v[13], 0, 0);
		mix(v[2], v[6], v[10], v[14], 0, 0);
		mix(v[3], v[7], v[11], v[15], 0, 0);
		mix(v[0], v[5], v[10], v[15], 0, 0);
		mix(v[1], v[6], v[11], v[12], 0, 0);
		mix(v[2], v[7], v[8], v[13], 0, 0);
		mix(v[3], v[4], v[9], v[14], 0, 0);
		// round 2
		mix(v[0], v[4], v[8], v[12], 0, 0);
		mix(v[1], v[5], v[9], v[13], 0, 0);
		mix(v[2], v[6], v[10], v[14], 0, 0);
		mix(v[3], v[7], v[11], v[15], 0, 0);
		mix(v[0], v[5], v[10], v[15], word1, 0);
		mix(v[1], v[6], v[11], v[12], 0, 0);
		mix(v[2], v[7], v[8], v[13], 0, 0);
		mix(v[3], v[4], v[9], v[14], 0, 0);
		// round 3
		mix(v[0], v[4], v[8], v[12], 0, 0);
		mix(v[1], v[5], v[9], v[13], 0, 0);
		mix(v[2], v[6], v[10], v[14], 0, 0);
		mix(v[3], v[7], v[11], v[15], 0, 0);
		mix(v[0], v[5], v[10], v[15], 0, 0);
		mix(v[1], v[6], v[11], v[12], 0, 0);
		mix(v[2], v[7], v[8], v[13], 0, word1);
		mix(v[3], v[4], v[9], v[14], 0, 0);
		// round 4
		mix(v[0], v[4], v[8], v[12], 0, 0);
		mix(v[1], v[5], v[9], v[13], 0, word1);
		mix(v[2], v[6], v[10], v[14], 0, 0);
		mix(v[3], v[7], v[11], v[15], 0, 0);
		mix(v[0], v[5], v[10], v[15], 0, 0);
		mix(v[1], v[6], v[11], v[12], 0, 0);
		mix(v[2], v[7], v[8], v[13], 0, 0);
		mix(v[3], v[4], v[9], v[14], 0, 0);
		// round 5
		mix(v[0], v[4], v[8], v[12], 0, 0);
		mix(v[1], v[5], v[9], v[13], 0, 0);
		mix(v[2], v[6], v[10], v[14], 0, 0);
		mix(v[3], v[7], v[11], v[15], 0, 0);
		mix(v[0], v[5], v[10], v[15], 0, word1);
		mix(v[1], v[6], v[11], v[12], 0, 0);
		mix(v[2], v[7], v[8], v[13], 0, 0);
		mix(v[3], v[4], v[9], v[14], 0, 0);
		// round 6
		mix(v[0], v[4], v[8], v[12], 0, 0);
		mix(v[1], v[5], v[9], v[13], 0, 0);
		mix(v[2], v[6], v[10], v[14], 0, 0);
		mix(v[3], v[7], v[11], v[15], 0, 0);
		mix(v[0], v[5], v[10], v[15], 0, 0);
		mix(v[1], v[6], v[11], v[12], 0, 0);
		mix(v[2], v[7], v[8], v[13], 0, 0);
		mix(v[3], v[4], v[9], v[14], word1, 0);
		// round 7
		mix(v[0], v[4], v[8], v[12], 0, 0);
		mix(v[1], v[5], v[9], v[13], word1, 0);
		mix(v[2], v[6], v[10], v[14], 0, 0);
		mix(v[3], v[7], v[11], v[15], 0, 0);
		mix(v[0], v[5], v[10], v[15], 0, 0);
		mix(v[1], v[6], v[11], v[12], 0, 0);
		mix(v[2], v[7], v[8], v[13], 0, 0);
		mix(v[3], v[4], v[9], v[14], 0, 0);
		// round 8
		mix(v[0], v[4], v[8], v[12], 0, 0);
		mix(v[1], v[5], v[9], v[13], 0, 0);
		mix(v[2], v[6], v[10], v[14], 0, word1);
		mix(v[3], v[7], v[11], v[15], 0, 0);
		mix(v[0], v[5], v[10], v[15], 0, 0);
		mix(v[1], v[6], v[11], v[12], 0, 0);
		mix(v[2], v[7], v[8], v[13], 0, 0);
		mix(v[3], v[4], v[9], v[14], 0, 0);
		// round 9
		mix(v[0], v[4], v[8], v[12], 0, 0);
		mix(v[1], v[5], v[9], v[13], 0, 0);
		mix(v[2], v[6], v[10], v[14], 0, 0);
		mix(v[3], v[7], v[11], v[15], 0, 0);
		mix(v[0], v[5], v[10], v[15], 0, 0);
		mix(v[1], v[6], v[11], v[12], 0, 0);
		mix(v[2], v[7], v[8], v[13], word1, 0);
		mix(v[3], v[4], v[9], v[14], 0, 0);
		// round 10
		mix(v[0], v[4], v[8], v[12], 0, 0);
		mix(v[1], v[5], v[9], v[13], 0, 0);
		mix(v[2], v[6], v[10], v[14], 0, 0);
		mix(v[3], v[7], v[11], v[15], word1, 0);
		mix(v[0], v[5], v[10], v[15], 0, 0);
		mix(v[1], v[6], v[11], v[12], 0, 0);
		mix(v[2], v[7], v[8], v[13], 0, 0);
		mix(v[3], v[4], v[9], v[14], 0, 0);
		// round 11
		mix(v[0], v[4], v[8], v[12], 0, word1);
		mix(v[1], v[5], v[9], v[13], 0, 0);
		mix(v[2], v[6], v[10], v[14], 0, 0);
		mix(v[3], v[7], v[11], v[15], 0, 0);
		mix(v[0], v[5], v[10], v[15], 0, 0);
		mix(v[1], v[6], v[11], v[12], 0, 0);
		mix(v[2], v[7], v[8], v[13], 0, 0);
		mix(v[3], v[4], v[9], v[14], 0, 0);
		// round 12
		mix(v[0], v[4], v[8], v[12], 0, 0);
		mix(v[1], v[5], v[9], v[13], 0, 0);
		mix(v[2], v[6], v[10], v[14], 0, 0);
		mix(v[3], v[7], v[11], v[15], 0, 0);
		mix(v[0], v[5], v[10], v[15], word1, 0);
		mix(v[1], v[6], v[11], v[12], 0, 0);
		mix(v[2], v[7], v[8], v[13], 0, 0);
		mix(v[3], v[4], v[9], v[14], 0, 0);

		// compress v into the blake state; this produces the 50-byte hash
		// (two Xi values)
		ulong h[7];
		h[0] = blake.h[0] ^ v[0] ^ v[8];
		h[1] = blake.h[1] ^ v[1] ^ v[9];
		h[2] = blake.h[2] ^ v[2] ^ v[10];
		h[3] = blake.h[3] ^ v[3] ^ v[11];
		h[4] = blake.h[4] ^ v[4] ^ v[12];
		h[5] = blake.h[5] ^ v[5] ^ v[13];
		h[6] = (blake.h[6] ^ v[6] ^ v[14]) & 0xffff;

		// store the two Xi values in the hash table
#if ZCASH_HASH_LEN == 50
		dropped += ht_store(0, ht, input * 2,
			h[0],
			h[1],
			h[2],
			h[3], rowCounter0);
		dropped += ht_store(0, ht, input * 2 + 1,
			(h[3] >> 8) | (h[4] << (64 - 8)),
			(h[4] >> 8) | (h[5] << (64 - 8)),
			(h[5] >> 8) | (h[6] << (64 - 8)),
			(h[6] >> 8), rowCounter0);
#else
#error "unsupported ZCASH_HASH_LEN"
#endif

		input++;
	}
#ifdef ENABLE_DEBUG
	debug[tid * 2] = 0;
	debug[tid * 2 + 1] = dropped;
#endif
}

#if NR_ROWS_LOG <= 16 && NR_SLOTS <= (1 << 8)

#define ENCODE_INPUTS(row, slot0, slot1) \
    ((row << 16) | ((slot1 & 0xff) << 8) | (slot0 & 0xff))
#define DECODE_ROW(REF)   (REF >> 16)
#define DECODE_SLOT1(REF) ((REF >> 8) & 0xff)
#define DECODE_SLOT0(REF) (REF & 0xff)

#elif NR_ROWS_LOG == 18 && NR_SLOTS <= (1 << 7)

#define ENCODE_INPUTS(row, slot0, slot1) \
    ((row << 14) | ((slot1 & 0x7f) << 7) | (slot0 & 0x7f))
#define DECODE_ROW(REF)   (REF >> 14)
#define DECODE_SLOT1(REF) ((REF >> 7) & 0x7f)
#define DECODE_SLOT0(REF) (REF & 0x7f)

#elif NR_ROWS_LOG == 19 && NR_SLOTS <= (1 << 6)

#define ENCODE_INPUTS(row, slot0, slot1) \
    ((row << 13) | ((slot1 & 0x3f) << 6) | (slot0 & 0x3f)) /* 1 spare bit */
#define DECODE_ROW(REF)   (REF >> 13)
#define DECODE_SLOT1(REF) ((REF >> 6) & 0x3f)
#define DECODE_SLOT0(REF) (REF & 0x3f)

#elif NR_ROWS_LOG == 20 && NR_SLOTS <= (1 << 6)

#define ENCODE_INPUTS(row, slot0, slot1) \
    ((row << 12) | ((slot1 & 0x3f) << 6) | (slot0 & 0x3f))
#define DECODE_ROW(REF)   (REF >> 12)
#define DECODE_SLOT1(REF) ((REF >> 6) & 0x3f)
#define DECODE_SLOT0(REF) (REF & 0x3f)

#else
#error "unsupported NR_ROWS_LOG"
#endif

/*
** Access a half-aligned long, that is a long aligned on a 4-byte boundary.
*/
__device__ ulong half_aligned_long(ulong *p, uint offset)
{
	return
		(((ulong)*(uint *)((char *)p + offset + 0)) << 0) |
		(((ulong)*(uint *)((char *)p + offset + 4)) << 32);
}

/*
** Access a well-aligned int.
*/
__device__ uint well_aligned_int(ulong *_p, uint offset)
{
	char *p = (char *)_p;
	return *(uint *)(p + offset);
}

/*
** XOR a pair of Xi values computed at "round - 1" and store the result in the
** hash table being built for "round". Note that when building the table for
** even rounds we need to skip 1 padding byte present in the "round - 1" table
** (the "0xAB" byte mentioned in the description at the top of this file.) But
** also note we can't load data directly past this byte because this would
** cause an unaligned memory access which is undefined per the OpenCL spec.
**
** Return 0 if successfully stored, or 1 if the row overflowed.
*/
__device__ uint xor_and_store(uint round, char *ht_dst, uint row,
	uint slot_a, uint slot_b, ulong *a, ulong *b,
	uint *rowCounters)
{
	ulong xi0, xi1, xi2;
#if NR_ROWS_LOG >= 16 && NR_ROWS_LOG <= 20
	// Note: for NR_ROWS_LOG == 20, for odd rounds, we could optimize by not
	// storing the byte containing bits from the previous PREFIX block for
	if (round == 1 || round == 2)
	{
		// xor 24 bytes
		xi0 = *(a++) ^ *(b++);
		xi1 = *(a++) ^ *(b++);
		xi2 = *a ^ *b;
		if (round == 2)
		{
			// skip padding byte
			xi0 = (xi0 >> 8) | (xi1 << (64 - 8));
			xi1 = (xi1 >> 8) | (xi2 << (64 - 8));
			xi2 = (xi2 >> 8);
		}
	}
	else if (round == 3)
	{
		// xor 20 bytes
		xi0 = half_aligned_long(a, 0) ^ half_aligned_long(b, 0);
		xi1 = half_aligned_long(a, 8) ^ half_aligned_long(b, 8);
		xi2 = well_aligned_int(a, 16) ^ well_aligned_int(b, 16);
	}
	else if (round == 4 || round == 5)
	{
		// xor 16 bytes
		xi0 = half_aligned_long(a, 0) ^ half_aligned_long(b, 0);
		xi1 = half_aligned_long(a, 8) ^ half_aligned_long(b, 8);
		xi2 = 0;
		if (round == 4)
		{
			// skip padding byte
			xi0 = (xi0 >> 8) | (xi1 << (64 - 8));
			xi1 = (xi1 >> 8);
		}
	}
	else if (round == 6)
	{
		// xor 12 bytes
		xi0 = *a++ ^ *b++;
		xi1 = *(uint *)a ^ *(uint *)b;
		xi2 = 0;
		if (round == 6)
		{
			// skip padding byte
			xi0 = (xi0 >> 8) | (xi1 << (64 - 8));
			xi1 = (xi1 >> 8);
		}
	}
	else if (round == 7 || round == 8)
	{
		// xor 8 bytes
		xi0 = half_aligned_long(a, 0) ^ half_aligned_long(b, 0);
		xi1 = 0;
		xi2 = 0;
		if (round == 8)
		{
			// skip padding byte
			xi0 = (xi0 >> 8);
		}
	}
	// invalid solutions (which start happenning in round 5) have duplicate
	// inputs and xor to zero, so discard them
	if (!xi0 && !xi1)
		return 0;
#else
#error "unsupported NR_ROWS_LOG"
#endif
	return ht_store(round, ht_dst, ENCODE_INPUTS(row, slot_a, slot_b),
		xi0, xi1, xi2, 0, rowCounters);
}

__device__ void equihash_round_cm3(uint round, char *ht_src, char *ht_dst, uint *rowCountersSrc, uint *rowCountersDst)
{
	uint                tid = blockIdx.x * blockDim.x + threadIdx.x;
	char				*p;
	uint				cnt;
	uint                i, j;
	uint				dropped_stor = 0;
	ulong				*a, *b;
	uint				xi_offset;
	xi_offset = (8 + ((round - 1) / 2) * 4);

	cnt = rowCountersSrc[tid];
	cnt = min(cnt, (uint)NR_SLOTS); // handle possible overflow in prev. round
	if (!cnt) {// no elements in row, no collisions
		return;
	}
	// find collisions
	p = (ht_src + tid * c_ROW_LEN) + xi_offset;
	for (i = 0; i < cnt; i++) {
		a = (ulong *)(p + i * 32);
		for (j = i + 1; j < cnt; j++) {
			b = (ulong *)(p + j * 32);
			dropped_stor += xor_and_store(round, ht_dst, tid, i, j, a, b, rowCountersDst);
		}
	}
}

#define KERNEL_ROUND_ODD_OLD(N) \
__global__  \
void kernel_round_cm3_ ## N( char *ht_src,  char *ht_dst) \
{ \
    equihash_round_cm3(N, ht_src, ht_dst, rowCounter0, rowCounter1); \
}


#define KERNEL_ROUND_EVEN_OLD(N) \
__global__  \
void kernel_round_cm3_ ## N(char *ht_src,  char *ht_dst) \
{ \
    equihash_round_cm3(N, ht_src, ht_dst, rowCounter1, rowCounter0); \
}


KERNEL_ROUND_ODD_OLD(1)
KERNEL_ROUND_EVEN_OLD(2)
KERNEL_ROUND_ODD_OLD(3)
KERNEL_ROUND_EVEN_OLD(4)
KERNEL_ROUND_ODD_OLD(5)
KERNEL_ROUND_EVEN_OLD(6)
KERNEL_ROUND_ODD_OLD(7)


__global__
void kernel_round_cm3_8(char *ht_src, char *ht_dst)
{
	uint tid = blockIdx.x * blockDim.x + threadIdx.x;
	equihash_round_cm3(8, ht_src, ht_dst, rowCounter1, rowCounter0);
	if (!tid) {
		sols.nr = sols.likely_invalids = 0;
	}
}


__device__ uint expand_ref(const char *ht, uint xi_offset, uint row, uint slot)
{
	return *(uint *)(ht + row * NR_SLOTS * SLOT_LEN + slot * SLOT_LEN + xi_offset - 4);
}

/*
** Expand references to inputs. Return 1 if so far the solution appears valid,
** or 0 otherwise (an invalid solution would be a solution with duplicate
** inputs, which can be detected at the last step: round == 0).
*/
__device__ uint expand_refs(uint *ins, uint nr_inputs, const char **htabs, uint round)
{
	const char	*ht = htabs[round & 1];
	uint		i = nr_inputs - 1;
	uint		j = nr_inputs * 2 - 1;
	uint		xi_offset = xi_offset_for_round(round);
	int			dup_to_watch = -1;
	do
	{
		ins[j] = expand_ref(ht, xi_offset,
			DECODE_ROW(ins[i]), DECODE_SLOT1(ins[i]));
		ins[j - 1] = expand_ref(ht, xi_offset,
			DECODE_ROW(ins[i]), DECODE_SLOT0(ins[i]));
		if (!round)
		{
			if (dup_to_watch == -1)
				dup_to_watch = ins[j];
			else if (ins[j] == dup_to_watch || ins[j - 1] == dup_to_watch)
				return 0;
		}
		if (!i)
			break;
		i--;
		j -= 2;
	} while (1);
	return 1;
}

/*
** Verify if a potential solution is in fact valid.
*/
__device__ void potential_sol(const char **htabs, uint ref0, uint ref1)
{
	uint	nr_values;
	uint	values_tmp[(1 << PARAM_K)];
	uint	sol_i;
	uint	i;
	nr_values = 0;
	values_tmp[nr_values++] = ref0;
	values_tmp[nr_values++] = ref1;
	uint round = PARAM_K - 1;
	do
	{
		round--;
		if (!expand_refs(values_tmp, nr_values, htabs, round))
			return;
		nr_values *= 2;
	} while (round > 0);
	// solution appears valid, copy it to sols
	sol_i = atomicAdd(&sols.nr, 1);
	if (sol_i >= MAX_SOLS)
		return;
	for (i = 0; i < (1 << PARAM_K); i++)
		sols.values[sol_i][i] = values_tmp[i];
	sols.valid[sol_i] = 1;
}

/*
** Scan the hash tables to find Equihash solutions.
*/
__global__
void kernel_sols(const char *ht0, const char *ht1)
{
	uint		tid = blockIdx.x * blockDim.x + threadIdx.x;
	const char	*htabs[2] = { ht0, ht1 };
	//uint	*hcounters[2] = { rowCounter0, rowCounter1 };
	uint		ht_i = (PARAM_K - 1) & 1; // table filled at last round
	uint		cnt;
	uint		xi_offset = xi_offset_for_round(PARAM_K - 1);
	uint		i, j;
	const char	*a, *b;
	uint		ref_i, ref_j;
	// it's ok for the collisions array to be so small, as if it fills up
	// the potential solutions are likely invalid (many duplicate inputs)
	ulong		collisions;
	//uint		coll;
#if NR_ROWS_LOG >= 16 && NR_ROWS_LOG <= 20
	// in the final hash table, we are looking for a match on both the bits
	// part of the previous PREFIX colliding bits, and the last PREFIX bits.
	uint		mask = 0xffffff;
#else
#error "unsupported NR_ROWS_LOG"
#endif

	a = htabs[ht_i] + tid * NR_SLOTS * SLOT_LEN;
	cnt = rowCounter0[tid];
	cnt = min(cnt, (uint)NR_SLOTS); // handle possible overflow in last round
	//coll = 0;
	a += xi_offset;
	for (i = 0; i < cnt; i++, a += SLOT_LEN) {
		uint a_data = ((*(uint *)a) & mask);
		ref_i = *(uint *)(a - 4);
		for (j = i + 1, b = a + SLOT_LEN; j < cnt; j++, b += SLOT_LEN) {
			if (a_data == ((*(uint *)b) & mask)) {
				ref_j = *(uint *)(b - 4);
				collisions = ((ulong)ref_i << 32) | ref_j;
				goto exit1;
			}
		}
	}
	return;

exit1:
	potential_sol(htabs, collisions >> 32, collisions & 0xffffffff);
}

static void sort_pair(uint32_t *a, uint32_t len)
{
	uint32_t    *b = a + len;
	uint32_t     tmp, need_sorting = 0;
	for (uint32_t i = 0; i < len; i++)
		if (need_sorting || a[i] > b[i])
		{
			need_sorting = 1;
			tmp = a[i];
			a[i] = b[i];
			b[i] = tmp;
		}
		else if (a[i] < b[i])
			return;
}

static uint32_t verify_sol(sols_t *sols, unsigned sol_i)
{
	uint32_t  *inputs = sols->values[sol_i];
	uint32_t  seen_len = (1 << (PREFIX + 1)) / 8;
	uint8_t seen[(1 << (PREFIX + 1)) / 8];
	uint32_t  i;
	uint8_t tmp;
	// look for duplicate inputs
	memset(seen, 0, seen_len);
	for (i = 0; i < (1 << PARAM_K); i++)
	{
		tmp = seen[inputs[i] / 8];
		seen[inputs[i] / 8] |= 1 << (inputs[i] & 7);
		if (tmp == seen[inputs[i] / 8])
		{
			// at least one input value is a duplicate
			sols->valid[sol_i] = 0;
			return 0;
		}
	}
	// the valid flag is already set by the GPU, but set it again because
	// I plan to change the GPU code to not set it
	sols->valid[sol_i] = 1;
	// sort the pairs in place
	for (uint32_t level = 0; level < PARAM_K; level++)
		for (i = 0; i < (1 << PARAM_K); i += (2 << level))
			sort_pair(&inputs[i], 1 << level);
	return 1;
}

struct __align__(64) c_context {
	char* buf_ht[2], *buf_dbg;
	sols_t	*sols;
	uint32_t nthreads;
	size_t global_ws;

	c_context(const uint32_t n_threads) {
		nthreads = n_threads;
	}
	void* operator new(size_t i) {
		return _mm_malloc(i, 64);
	}
	void operator delete(void* p) {
		_mm_free(p);
	}
};

static void compress(uint8_t *out, uint32_t *inputs, uint32_t n)
{
	uint32_t byte_pos = 0;
	int32_t bits_left = PREFIX + 1;
	uint8_t x = 0;
	uint8_t x_bits_used = 0;
	uint8_t *pOut = out;
	while (byte_pos < n)
	{
		if (bits_left >= 8 - x_bits_used)
		{
			x |= inputs[byte_pos] >> (bits_left - 8 + x_bits_used);
			bits_left -= 8 - x_bits_used;
			x_bits_used = 8;
		}
		else if (bits_left > 0)
		{
			uint32_t mask = ~(-1 << (8 - x_bits_used));
			mask = ((~mask) >> bits_left) & mask;
			x |= (inputs[byte_pos] << (8 - x_bits_used - bits_left)) & mask;
			x_bits_used += bits_left;
			bits_left = 0;
		}
		else if (bits_left <= 0)
		{
			assert(!bits_left);
			byte_pos++;
			bits_left = PREFIX + 1;
		}
		if (x_bits_used == 8)
		{
			*pOut++ = x;
			x = x_bits_used = 0;
		}
	}
}

sa_cuda_context::sa_cuda_context(int tpb, int blocks, int id)
	: threadsperblock(tpb), totalblocks(blocks), device_id(id)
{
	checkCudaErrors(hipSetDevice(device_id));
	checkCudaErrors(hipDeviceReset());
	checkCudaErrors(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));
	checkCudaErrors(hipDeviceSetCacheConfig(hipFuncCachePreferShared));

	eq = new c_context(threadsperblock * totalblocks);
#ifdef ENABLE_DEBUG
	size_t              dbg_size = NR_ROWS;
#else
	size_t              dbg_size = 1;
#endif

	checkCudaErrors(hipMalloc((void**)&eq->buf_dbg, dbg_size));
	checkCudaErrors(hipMalloc((void**)&eq->buf_ht[0], HT_SIZE));
	checkCudaErrors(hipMalloc((void**)&eq->buf_ht[1], HT_SIZE));
	checkCudaErrors(hipHostMalloc(&eq->sols, sizeof(*eq->sols)));
	checkCudaErrors(hipDeviceSynchronize());
}

sa_cuda_context::~sa_cuda_context()
{
	checkCudaErrors(hipSetDevice(device_id));
	checkCudaErrors(hipDeviceReset());
	delete eq;
}

void sa_cuda_context::solve(const char * tequihash_header, unsigned int tequihash_header_len, const char * nonce, unsigned int nonce_len, std::function<bool()> cancelf, std::function<void(const std::vector<uint32_t>&, size_t, const unsigned char*)> solutionf, std::function<void(void)> hashdonef)
{
	checkCudaErrors(hipSetDevice(device_id));

	unsigned char context[140];
	memset(context, 0, 140);
	memcpy(context, tequihash_header, tequihash_header_len);
	memcpy(context + tequihash_header_len, nonce, nonce_len);

	c_context *miner = eq;

	//FUNCTION<<<totalblocks, threadsperblock>>>(ARGUMENTS)

	blake2b_state_t initialCtx;
	zcash_blake2b_init(&initialCtx, ZCASH_HASH_LEN, PARAM_N, PARAM_K);
	zcash_blake2b_update(&initialCtx, (const uint8_t*)context, 128, 0);

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(blake), &initialCtx, sizeof(blake2b_state_s), 0, hipMemcpyHostToDevice));

	constexpr uint32_t THREAD_SHIFT = 7;
	constexpr uint32_t THREAD_COUNT = 1 << THREAD_SHIFT;
	constexpr uint32_t DIM_SIZE = NR_ROWS >> THREAD_SHIFT;

	kernel_init_0 << <DIM_SIZE, THREAD_COUNT >> > ();
	kernel_round0<<<1024, 64>>> (miner->buf_ht[0], 16);
	if (cancelf()) return;
	kernel_init_1 << <DIM_SIZE, THREAD_COUNT >> > ();
	kernel_round_cm3_1 << < DIM_SIZE, THREAD_COUNT >> > (miner->buf_ht[0], miner->buf_ht[1]);
	if (cancelf()) return;
	kernel_init_0 << <DIM_SIZE, THREAD_COUNT >> > ();
	kernel_round_cm3_2 << < DIM_SIZE, THREAD_COUNT >> > (miner->buf_ht[1], miner->buf_ht[0]);
	if (cancelf()) return;
	kernel_init_1 << <DIM_SIZE, THREAD_COUNT >> > ();
	kernel_round_cm3_3 << <DIM_SIZE, THREAD_COUNT >> > (miner->buf_ht[0], miner->buf_ht[1]);
	if (cancelf()) return;
	kernel_init_0 << <DIM_SIZE, THREAD_COUNT >> > ();
	kernel_round_cm3_4 << < DIM_SIZE, THREAD_COUNT >> > (miner->buf_ht[1], miner->buf_ht[0]);
	if (cancelf()) return;
	kernel_init_1 << <DIM_SIZE, THREAD_COUNT >> > ();
	kernel_round_cm3_5 << < DIM_SIZE, THREAD_COUNT >> > (miner->buf_ht[0], miner->buf_ht[1]);
	if (cancelf()) return;
	kernel_init_0 << <DIM_SIZE, THREAD_COUNT >> > ();
	kernel_round_cm3_6 << < DIM_SIZE, THREAD_COUNT >> > (miner->buf_ht[1], miner->buf_ht[0]);
	if (cancelf()) return;
	kernel_init_1 << <DIM_SIZE, THREAD_COUNT >> > ();
	kernel_round_cm3_7 << < DIM_SIZE, THREAD_COUNT >> > (miner->buf_ht[0], miner->buf_ht[1]);
	if (cancelf()) return;
	kernel_init_0 << <DIM_SIZE, THREAD_COUNT >> > ();
	kernel_round_cm3_8 << < DIM_SIZE, THREAD_COUNT >> > (miner->buf_ht[1], miner->buf_ht[0]);
	if (cancelf()) return;
	kernel_sols << < DIM_SIZE, THREAD_COUNT >> > (miner->buf_ht[0], miner->buf_ht[1]);

	checkCudaErrors(hipMemcpyFromSymbol(miner->sols, HIP_SYMBOL(sols), sizeof(sols_t), 0, hipMemcpyDeviceToHost));

	if (miner->sols->nr > MAX_SOLS)
		miner->sols->nr = MAX_SOLS;

	for (unsigned sol_i = 0; sol_i < miner->sols->nr; sol_i++) {
		verify_sol(miner->sols, sol_i);
	}


	uint8_t proof[COMPRESSED_PROOFSIZE * 2];
	for (uint32_t i = 0; i <  miner->sols->nr; i++) {
		if (miner->sols->valid[i]) {
			compress(proof, (uint32_t *)(miner->sols->values[i]), 1 << PARAM_K);
			solutionf(std::vector<uint32_t>(0), 1344, proof);
		}
	}
	hashdonef();

}